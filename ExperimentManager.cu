#include "hip/hip_runtime.h"
/*
 * ExperimentManager.cpp
 *
 *  Created on: 23/06/2019
 *      Author: pablomoreira
 */

#include "ExperimentManager.h"
#include <chrono>
#include <thread>
#include <string>
#include <hip/hip_runtime_api.h>

const array<string,4> ExperimentManager::DISPONIBLE_FUNCTIONS = {"dtrsm","dgemm","dsyrk","dpotrf"};
const int ExperimentManager::STANDARD_SIZE = 512;
const int ExperimentManager::STANDARD_ATTEMPTS = 10;


ExperimentManager::ExperimentManager(int inputSize,int batches, bool concurrent, string func1, string func2)
{

	m_concurrent = concurrent;
	m_batchesNumber = batches;
	m_inputSize = inputSize;


	m_input1 = thrust::device_vector<double> (m_inputSize * m_inputSize);
	m_input2 = thrust::device_vector<double> (m_inputSize * m_inputSize);

	m_func1OutputVectors = vector<thrust::device_vector<double>>(m_batchesNumber);
	m_func2OutputVectors = vector<thrust::device_vector<double>>(m_batchesNumber);

	GPU_fill_rand(thrust::raw_pointer_cast(&m_input1[0]), m_inputSize, m_inputSize);
	GPU_fill_rand(thrust::raw_pointer_cast(&m_input2[0]), m_inputSize, m_inputSize);

	func1cuSolver = func2cuSolver = false;
	for(int i = 0; i < DISPONIBLE_FUNCTIONS.size(); ++i)
	{
		if(func1 == DISPONIBLE_FUNCTIONS[i])
		{
			m_func1 = i;
			if(i == 3)
			{
				func1cuSolver = true;

				hipsolverHandle_t handlePotrf;
				hipsolverDnCreate(&handlePotrf);
				hipsolverDnDpotrf_bufferSize(handlePotrf,HIPBLAS_FILL_MODE_UPPER,m_inputSize,thrust::raw_pointer_cast(&m_input1[0]),m_inputSize,&m_func1cusolverWorkspaceSize );
			}
		}
		if(func2 == DISPONIBLE_FUNCTIONS[i])
		{
			m_func2 = i;
			if(i == 3)
			{
				func2cuSolver = true;

				hipsolverHandle_t handlePotrf;
				hipsolverDnCreate(&handlePotrf);
				hipsolverDnDpotrf_bufferSize(handlePotrf,HIPBLAS_FILL_MODE_UPPER,m_inputSize,thrust::raw_pointer_cast(&m_input2[0]),m_inputSize,&m_func2cusolverWorkspaceSize );
			}
		}

	}

	if(!func1cuSolver)
		func1Handles = vector<hipblasHandle_t>(m_batchesNumber);
	else
		func1cuSolverHandles = vector<hipsolverHandle_t>(m_batchesNumber);

	if(!func2cuSolver)
		func2Handles = vector<hipblasHandle_t>(m_batchesNumber);
	else
		func2cuSolverHandles = vector<hipsolverHandle_t>(m_batchesNumber);

	if(m_concurrent)
	{
		func1Streams = vector<hipStream_t>(m_batchesNumber);
		func2Streams = vector<hipStream_t>(m_batchesNumber);
	}

	for(int i=0; i<m_batchesNumber; i++)
	{

		if(!func1cuSolver)
		{
			hipblasCreate(&func1Handles[i]);
			m_func1OutputVectors[i] = thrust::device_vector<double>(m_inputSize * m_inputSize);
		}
		else
		{
			hipsolverDnCreate(&func1cuSolverHandles[i]);
			m_func1OutputVectors[i] = thrust::device_vector<double>(m_func1cusolverWorkspaceSize);
		}

		if(!func2cuSolver)
		{
			hipblasCreate(&func2Handles[i]);
			m_func2OutputVectors[i] = thrust::device_vector<double>(m_inputSize * m_inputSize);
		}
		else
		{
			hipsolverDnCreate(&func2cuSolverHandles[i]);
			m_func2OutputVectors[i] = thrust::device_vector<double>(m_func2cusolverWorkspaceSize);
		}




	}

//	if(!func2cuSolver)
//	{
//		m_func2OutputVectors = thrust::device_vector<double>(m_inputSize * m_inputSize * m_batchesNumber);
//	}
//	else
//	{
//		m_func2OutputVectors = thrust::device_vector<double>(m_func2cusolverWorkspaceSize * m_batchesNumber);
//	}

}

ExperimentManager::~ExperimentManager() {
	// TODO Auto-generated destructor stub



}

void ExperimentManager::GPU_fill_rand(double *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniformDouble(prng, A, nr_rows_A * nr_cols_A);
}

void ExperimentManager::runExperiments(){


	int *potrfstates1;
	int *potrfstates2;

	if(m_func1 ==3)
		potrfstates1 = new int[m_batchesNumber];
	if(m_func2 ==3)
		potrfstates2 = new int[m_batchesNumber];

	for(int i=0; i<m_batchesNumber; i++)
	{

		if(m_concurrent)
		{
			hipStreamCreate(&func1Streams[i]);
			hipStreamCreate(&func2Streams[i]);
			if(!func1cuSolver)
				hipblasSetStream(func1Handles[i], func1Streams[i]);
			else
				hipsolverSetStream(func1cuSolverHandles[i], func1Streams[i]);
			if(!func2cuSolver)
				hipblasSetStream(func2Handles[i], func2Streams[i]);
			else
				hipsolverSetStream(func2cuSolverHandles[i], func2Streams[i]);
		}

		string func1 = DISPONIBLE_FUNCTIONS[m_func1];
		string func2 = DISPONIBLE_FUNCTIONS[m_func2];

		switch(m_func1)
		{
			case 0:
				gpu_blas_dtrsm(&func1Handles[i],thrust::raw_pointer_cast(&m_input1[0]), thrust::raw_pointer_cast(&m_func1OutputVectors[i][0]), m_inputSize, m_inputSize);
				break;
			case 1:
				gpu_blas_mmul(&func1Handles[i],thrust::raw_pointer_cast(&m_input1[0]), thrust::raw_pointer_cast(&m_input2[0]), thrust::raw_pointer_cast(&m_func1OutputVectors[i][0]), m_inputSize, m_inputSize, m_inputSize);
				break;
			case 2:
				gpu_blas_dsyrk(&func1Handles[i],thrust::raw_pointer_cast(&m_input1[0]),  thrust::raw_pointer_cast(&m_func1OutputVectors[i][0]), m_inputSize, m_inputSize);
				break;
			case 3:
				gpu_solv_dpotrf(&func1cuSolverHandles[i],thrust::raw_pointer_cast(&m_input1[0]),  thrust::raw_pointer_cast(&m_func1OutputVectors[i][0]), m_inputSize, m_func1cusolverWorkspaceSize, &potrfstates1[i]);
				break;

		}

		switch(m_func2)
		{
			case 0:
				gpu_blas_dtrsm(&func2Handles[i],thrust::raw_pointer_cast(&m_input1[0]), thrust::raw_pointer_cast(&m_func2OutputVectors[i][0]), m_inputSize, m_inputSize);
				break;
			case 1:
				gpu_blas_mmul(&func2Handles[i],thrust::raw_pointer_cast(&m_input1[0]), thrust::raw_pointer_cast(&m_input2[0]), thrust::raw_pointer_cast(&m_func2OutputVectors[i][0]), m_inputSize, m_inputSize, m_inputSize);
				break;
			case 2:
				gpu_blas_dsyrk(&func2Handles[i],thrust::raw_pointer_cast(&m_input1[0]),  thrust::raw_pointer_cast(&m_func2OutputVectors[i][0]), m_inputSize, m_inputSize);
				break;
			case 3:
				gpu_solv_dpotrf(&func2cuSolverHandles[i],thrust::raw_pointer_cast(&m_input1[0]),  thrust::raw_pointer_cast(&m_func2OutputVectors[i][0]), m_inputSize, m_func2cusolverWorkspaceSize, &potrfstates2[i]);
				break;

		}







	}

	hipDeviceSynchronize();

//	std::this_thread::sleep_for (std::chrono::seconds(10));




	cout << "finished" <<endl;
}

void ExperimentManager::gpu_blas_mmul(hipblasHandle_t* handle, const double *A, const double *B, double *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;

	hipblasDgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);


}

void ExperimentManager::gpu_blas_dsyrk(hipblasHandle_t* handle,const double *A, double *C, const int n, const int k) {
	int lda=k,ldc=k;
	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;

	hipblasDsyrk(*handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, n, k, alpha, A,lda, beta, C, ldc);

}

void ExperimentManager::gpu_solv_dpotrf(hipsolverHandle_t* handle,double *A, double* W,const int n, const int k, int* devInfo) {

	int lda=n;

	hipsolverDnDpotrf(*handle, HIPBLAS_FILL_MODE_UPPER, n, A, lda, W, k, devInfo);

}

void ExperimentManager::gpu_blas_dtrsm(hipblasHandle_t* handle, double *A, double *B, int a_b_rows, int a_b_cols){

	int lda=a_b_rows,ldb=a_b_cols;
	const double alf = 1;
	const double *alpha = &alf;


	hipblasDtrsm(*handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, a_b_rows, a_b_cols, alpha, A, lda, B, ldb);




}
