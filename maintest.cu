#include "hip/hip_runtime.h"
// High level matrix multiplication on GPU using CUDA with Thrust, CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>
#include <array>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hipsolver.h>


// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(double *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniformDouble(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(hipblasHandle_t* handle, const double *A, const double *B, double *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;

	// Create a handle for CUBLAS
	// hipblasHandle_t handle;
	// hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasDgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	// hipblasDestroy(*handle);
}

void gpu_blas_dsyrk(hipblasHandle_t* handle,const double *A, double *C, const int n, const int k) {
	int lda=k,ldc=k;
	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;

	// Create a handle for CUBLAS
	// hipblasHandle_t handle;
	// hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasDsyrk(*handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, n, k, alpha, A,lda, beta, C, ldc);

	// Destroy the handle
	// hipblasDestroy(*handle);
}

void gpu_solv_dpotrf(hipsolverHandle_t* handle,double *A, double* W,const int n, const int k, int* devInfo) {
	int lda=n;

	// Create a handle for CUBLAS
	// hipblasHandle_t handle;
	// hipblasCreate(&handle);

	// Do the actual multiplication

	hipsolverDnDpotrf(*handle, HIPBLAS_FILL_MODE_UPPER, n, A, lda, W, k, devInfo);


	// Destroy the handle
	// hipsolverDnDestroy(*handle);
}

void gpu_blas_dtrsm(hipblasHandle_t* handle, double *A, double *B, int a_b_rows, int a_b_cols){

	int lda=a_b_rows,ldb=a_b_cols;
	const double alf = 1;
	const double *alpha = &alf;


	hipblasDtrsm(*handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, a_b_rows, a_b_cols, alpha, A, lda, B, ldb);

	// Destroy the handle
	hipblasDestroy(*handle);


}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const thrust::device_vector<double> &A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

//int main() {
//
//    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
//
//	// for simplicity we are going to use square arrays
//	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 512;
//
//	thrust::device_vector<double> d_A(nr_rows_A * nr_cols_A), d_B(nr_rows_B * nr_cols_B), d_C(nr_rows_C * nr_cols_C);
//
//	// Fill the Input arrays
//	GPU_fill_rand(thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
//    GPU_fill_rand(thrust::raw_pointer_cast(&d_B[0]), nr_rows_B, nr_cols_B);
//
//    // bool potrfMode = false;
//
//    int batch_count = 10;
//
//    std::array<thrust::device_vector<double>,10> answerVectors;
//	std::array<thrust::device_vector<double>,10> answerVectorsB;
//
//	std::array<hipblasHandle_t, 10> handles;
//	std::array<hipStream_t, 10> streams;
//
//	std::array<hipblasHandle_t, 10> handlesB;
//    std::array<hipStream_t, 10> streamsB;
//
//    // std::array<hipsolverHandle_t, 10> handlesPotrf;
//    // std::array<hipStream_t, 10> streamsPotrf;
//    // std::array<int, 10> potrfAns;
//    // hipStream_t *streams = (hipStream_t *) malloc(batch_count*sizeof(hipStream_t));
//
//    // hipsolverHandle_t handlePotrf;
//    // hipsolverDnCreate(&handlePotrf);
//    // int workspaceSize = -1;
//    // hipsolverDnDpotrf_bufferSize(handlePotrf,HIPBLAS_FILL_MODE_UPPER,nr_rows_A,thrust::raw_pointer_cast(&d_A[0]),nr_rows_A,&workspaceSize );
//    // nr_rows_C = nr_cols_C = workspaceSize;
//	for(int i=0; i<batch_count; i++)
//	{
//		std::cout << "batch " << i << " initializing"<<std::endl;
//
//		hipStreamCreate(&streams[i]);
//		hipblasCreate(&handles[i]);
//
//		hipStreamCreate(&streamsB[i]);
//		hipblasCreate(&handlesB[i]);
//
//		hipblasSetStream(handles[i], streams[i]);
//
//        hipblasSetStream(handlesB[i], streamsB[i]);
//
//        // hipStreamCreate(&streamsPotrf[i]);
//        // hipsolverDnCreate(&handlesPotrf[i]);
//        // hipsolverSetStream(handlesPotrf[i], streamsPotrf[i]);
//
//        thrust::device_vector<double> d_C(nr_rows_C * nr_cols_C);
//        thrust::device_vector<double> d_C2(nr_rows_C * nr_cols_C);
//        // thrust::device_vector<double> d_C2(workspaceSize);
//
//		answerVectors[i] = d_C;
//		answerVectorsB[i] = d_C2;
//	}
//
//
//	for(int i=0; i<batch_count; i++){
//		// Set CUDA stream
//
//		std::cout << "passou aqui" << std::endl;
//
//		// DGEMM: C = alpha*A*B + beta*C
//		gpu_blas_mmul(&handles[i],thrust::raw_pointer_cast(&d_A[0]), thrust::raw_pointer_cast(&d_B[0]), thrust::raw_pointer_cast(&answerVectors[i][0]), nr_rows_A, nr_cols_A, nr_cols_B);
//
//        // gpu_blas_dsyrk(&handlesB[i],thrust::raw_pointer_cast(&d_A[0]),  thrust::raw_pointer_cast(&answerVectorsB[i][0]), nr_rows_A, nr_cols_C);
//
//		// gpu_solv_dpotrf(&handlesPotrf[i],thrust::raw_pointer_cast(&d_A[0]),  thrust::raw_pointer_cast(&answerVectorsB[i][0]), nr_rows_A, workspaceSize, &potrfAns[i]);
//
//		gpu_blas_dtrsm(&handlesB[i],thrust::raw_pointer_cast(&d_A[0]), thrust::raw_pointer_cast(&d_B[0]), nr_rows_A, nr_cols_B);
//	}
//
//
//
//
//	std::cout << "finished" <<std::endl;
//
//	return 0;
//}
